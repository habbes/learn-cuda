
#include <hip/hip_runtime.h>
#include <stdio.h>

hipError_t checkCudaError(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(result));
    }
    return result;
}

void initWith(float num, float *a, int N)
{
    for (int i = 0; i < N; ++i)
    {
        a[i] = num;
    }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int stride = gridDim.x * blockDim.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *array, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (array[i] != target)
        {
            printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
            exit(1);
        }
    }
    printf("SUCCESS! All values added correctly.\n");
}

int main()
{
    const int N = 2 << 20;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;

    checkCudaError(hipMallocManaged(&a, size));
    checkCudaError(hipMallocManaged(&b, size));
    checkCudaError(hipMallocManaged(&c, size));

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    size_t threads_per_block = 256;
    size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);
    checkCudaError(hipGetLastError());

    checkCudaError(hipDeviceSynchronize());

    checkElementsAre(7, c, N);

    checkCudaError(hipFree(a));
    checkCudaError(hipFree(b));
    checkCudaError(hipFree(c));
}
